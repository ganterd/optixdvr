#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

// optix code:
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "prd.h"
#include "sampling.h"

/*! the 'builtin' launch index we need to render a frame */
rtDeclareVariable(uint2, pixelID,   rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim,   );

/*! the ray related state */
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData, prd, rtPayload, );

/*! the 2D, float3-type color frame buffer we'll write into */
rtBuffer<uchar4, 2> fb;

rtDeclareVariable(int, numSamples, , );
rtDeclareVariable(int, maxBounces, , );
rtDeclareVariable(float, ertThreshold, , );
rtDeclareVariable(int, highlightERT, , );
rtDeclareVariable(int, showDepthComplexity, , );
rtDeclareVariable(int, showPageTableAccesses, , );

rtDeclareVariable(rtObject, world, , );

rtDeclareVariable(float3, camera_lower_left_corner, , );
rtDeclareVariable(float3, camera_horizontal, , );
rtDeclareVariable(float3, camera_vertical, , );
rtDeclareVariable(float3, camera_origin, , );
rtDeclareVariable(float3, camera_u, , );
rtDeclareVariable(float3, camera_v, , );
rtDeclareVariable(float, camera_lens_radius, , );

rtDeclareVariable(float3, volumeDimensions, , );
rtDeclareVariable(float3, volumeMin, , );
rtDeclareVariable(float3, volumeSize, , );
rtDeclareVariable(float3, poolDimensions, , );

struct Camera {
  static __device__ optix::Ray generateRay(float s, float t, DRand48 &rnd)
  {
    const vec3f rd = camera_lens_radius * random_in_unit_disk(rnd);
    const vec3f lens_offset = camera_u * rd.x + camera_v * rd.y;
    const vec3f origin = camera_origin + lens_offset;
    const vec3f direction
      = camera_lower_left_corner
      + s * camera_horizontal
      + t * camera_vertical
      - origin;
    return optix::make_Ray(origin.as_float3(),
                          normalize(direction).as_float3(), 0, 1e-6f, RT_DEFAULT_MAX);
  }
};

inline __device__ vec4f missColor(const optix::Ray &ray)
{
  //const vec3f unit_direction = normalize(ray.direction);
  //const float t = 0.5f*(unit_direction.y + 1.0f);
  //const vec3f c = (1.0f - t)*vec3f(1.0f, 1.0f, 1.0f) + t * vec3f(0.5f, 0.7f, 1.0f);
  return vec4f(0.0f, 0.0f, 0.0f, 0.0f);
}

bool __device__ entryExitPoints(
  optix::Ray &ray,
  const vec3f& boxmin,
  const vec3f& boxmax,
  float &entryDistance,
  float &exitDistance
){
    const vec3f rayDirectionInverse = 1.0f / ray.direction;
    vec3f vminv = (boxmin - ray.origin) * rayDirectionInverse;
    vec3f vmaxv = (boxmax - ray.origin) * rayDirectionInverse;

    vec3f vmin = min(vminv, vmaxv);
    vec3f vmax = max(vminv, vmaxv);

    float tmin = fmaxf(fmaxf(vmin.x, vmin.y), vmin.z);
    float tmax = fminf(fminf(vmax.x, vmax.y), vmax.z);

    tmin = fmaxf(ray.tmin + 1e-9f, tmin);

    entryDistance = tmin;
    exitDistance = tmax;
    return tmax > tmin;
}

inline __device__ vec4f color(optix::Ray &ray, DRand48 &rnd)
{
  PerRayData prd;

  vec4f accumulatedColour(vec3f(0.0f), 0.0f);
  prd.out.accumulation = accumulatedColour;

  float volumeEntryDistance, volumeExitDistance;
  vec3f boxsize = volumeSize;
  vec3f boxmin = volumeMin;
  vec3f boxmax = volumeMin + volumeSize;
  bool intersect = entryExitPoints(ray, boxmin, boxmax, volumeEntryDistance, volumeExitDistance);
  if(!intersect)
   return accumulatedColour;

  float worldSpaceDepth = volumeExitDistance - volumeEntryDistance;

  vec3f volumeEntryPoint = ray.origin + ray.direction * volumeEntryDistance;
  vec3f volumeExitPoint = ray.origin + ray.direction * volumeExitDistance;
  volumeEntryPoint = (volumeEntryPoint + boxmax) / boxsize;
  volumeExitPoint = (volumeExitPoint + boxmax) / boxsize;

  vec3f volumeDirection = volumeExitPoint - volumeEntryPoint;
  float volumeSpaceDepth = volumeDirection.length();

  vec3f dataSpaceVector = volumeDirection * volumeDimensions;
  vec3f poolSpaceStep = normalize(dataSpaceVector) / (2.0f * vec3f(poolDimensions));

  float steps = (2.0f * dataSpaceVector.length());
  vec3f volumeSpaceStep = normalize(volumeDirection) / (2.0f * volumeDimensions);
  float volumeSpaceStepSize = volumeSpaceStep.length();
  float worldSpaceStepSize = worldSpaceDepth / steps;//volumeSpaceStepSize * (worldSpaceDepth /volumeSpaceDepth);

  prd.in.rayDirectionInverse = make_float3(1.0f, 1.0f, 1.0f) / ray.direction;
  prd.in.worldSpaceStepSize = worldSpaceStepSize;
  prd.in.worldSpaceStepSizeInv = 1.0f / worldSpaceStepSize;
  prd.in.poolSpaceStep = poolSpaceStep;
  prd.in.opacityCorrection = volumeSpaceStepSize * 150.0f;
  prd.in.volumeSpaceStep = volumeSpaceStep;
  prd.out.pageTableAccesses = 0;

  //prd.out.accumulation = vec4f(volumeEntryPoint, 1.0f);

  prd.rayTerminated = false;

  int depth = 0;
  for (; depth < maxBounces; ++depth) {
    rtTrace(world, ray, prd);

    if(prd.out.accumulation.w >= ertThreshold || prd.rayTerminated)
      break;

    ray.tmin = prd.hit.exitDistance;
  }

  if(showDepthComplexity)
  {
    /* Convert scalar to RGB heat (blue minimum, red maximum) */
    float minimum = 0.0f;
    float maximum = (float)maxBounces;
    float ratio = 2 * ((float)depth - minimum) / (maximum - minimum);
    float b = fmaxf(0.0f, (1.0f - ratio));
    float r = fmaxf(0.0f, (ratio - 1.0f));
    float g = 1.0f - b - r;
    prd.out.accumulation = vec4f(r,g,b, 1.0f);
  }
  else if(showPageTableAccesses)
  {
    float minimum = 0.0f;
    float maximum = (float)maxBounces;
    float ratio = 2 * ((float)prd.out.pageTableAccesses - minimum) / (maximum - minimum);
    float b = fmaxf(0.0f, (1.0f - ratio));
    float r = fmaxf(0.0f, (ratio - 1.0f));
    float g = 1.0f - b - r;
    prd.out.accumulation = vec4f(r,g,b, 1.0f);
  }
  else
  {
    /* Highlight the ERT-terminated pixels */
    if(highlightERT)
    {
      if(prd.out.accumulation.w >= ertThreshold)
      {
        /* Highlight the ERT with red */
        prd.out.accumulation = vec4f(1, 0, 0, 1);
      }
      else
      {
        /* Convert anything that isn't an ERT to grayscale */
        vec4f c = prd.out.accumulation;
        float luminance = c.x * 0.2126f + c.y * 0.7152f + c.z * 0.0722f;
        prd.out.accumulation = vec4f(vec3f(luminance), 1.0f);
      }
    }
  }

  return prd.out.accumulation;
}

/*! the actual ray generation program - note this has no formal
  function parameters, but gets its paramters throught the 'pixelID'
  and 'pixelBuffer' variables/buffers declared above */
RT_PROGRAM void renderPixel()
{
  int pixel_index = pixelID.y * launchDim.x + pixelID.x;
  vec4f col(0.0f);
  DRand48 rnd;
  rnd.init(pixel_index);
  for (int s = 0; s < numSamples; s++) {
    float u = float(pixelID.x) / float(launchDim.x);
    float v = float(pixelID.y) / float(launchDim.y);
    optix::Ray ray = Camera::generateRay(u, v, rnd);
    col += color(ray, rnd);
  }
  col = col / float(numSamples);

  col = saturate(col);
  uchar4 c;
  c.x = col.x * 255.0f;
  c.y = col.y * 255.0f;
  c.z = col.z * 255.0f;
  //c.w = col.w * 255.0f;
  c.w = 255.0f;
  fb[pixelID] = c;
}
