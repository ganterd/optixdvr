#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "prd.h"
#include "brickpoolentry.h"

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData, prd, rtPayload, );

rtDeclareVariable(int, dontSample, , );
rtTextureSampler<float, 3> volumeTexture;
rtTextureSampler<float4, 1> transferFunction;
rtDeclareVariable(float3, brickSizeVolumeSpace, , );
rtDeclareVariable(float3, volumeMin, , );
rtDeclareVariable(float3, volumeSize, , );

inline __device__ void accumulate(
    const optix::Ray &ray_in,
    float entryDistance,
    float exitDistance,
    vec4f &accumulation
){
    const vec3f rayDirection = vec3f(ray.direction);
    const vec3f rayOrigin = vec3f(ray.origin);

    /* Round up and down to the nearest entry/exit samples */
    int startSample = (int)ceilf(entryDistance / prd.in.worldSpaceStepSize);
    int endSample = (int)floorf(exitDistance / prd.in.worldSpaceStepSize);
    exitDistance = (float)endSample * prd.in.worldSpaceStepSize;
    prd.hit.exitDistance = exitDistance + prd.in.worldSpaceStepSize;
    const int steps = endSample - startSample;

    /* Get the world space entry point */
    const vec3f worldSpaceEntry = rayDirection * entryDistance + rayOrigin;

    /* Convert this to volume space */
    const vec3f brickEntryPoint = (worldSpaceEntry - volumeMin) / volumeSize;

    /* Ray-stepping loop */
    vec3f p = brickEntryPoint;
    vec4f a = accumulation;
    const vec3f step = prd.in.volumeSpaceStep;
    const float opacityCorrection = prd.in.opacityCorrection;
    vec3f pageTableIndex;
    vec3f prevPageTableIndex(-1.0f);
    vec3f poolOffset;
    vec3f brickBegin;
    const vec3f brickSizeInv = vec3f(1.0f) / brickSizeVolumeSpace;
    int ptaccesses = 0;
    for(int i = 0; i < steps && a.w < 0.99f; ++i)
    {

        pageTableIndex.x = floorf(p.x * brickSizeInv.x);
        pageTableIndex.y = floorf(p.y * brickSizeInv.y);
        pageTableIndex.z = floorf(p.z * brickSizeInv.z);

        /* If we've moved to a new brick, need to fetch page table info */
        if(pageTableIndex != prevPageTableIndex)
        {
            ushort4 pageTableEntry = tex3D(pageTableTexture, pageTableIndex.x, pageTableIndex.y, pageTableIndex.z);
            if(pageTableEntry.w == PageTableEntryNotPaged)
            {
                p += step;
                continue;
            }
            brickBegin = pageTableIndex * brickSizeVolumeSpace;

            /* For some reason, have to push the offset half a voxel for visual fix? */
            poolOffset.x = (float)pageTableEntry.x * poolDataRegionSize.x + 0.5f;
            poolOffset.y = (float)pageTableEntry.y * poolDataRegionSize.y + 0.5f;
            poolOffset.z = (float)pageTableEntry.z * poolDataRegionSize.z + 0.5f;
            prevPageTableIndex = pageTableIndex;
            ptaccesses++;
        }

        /* Convert p from normalized volume space to pool data space */
        vec3f voxelAddress;
        voxelAddress.x = poolOffset.x + ((p.x - brickBegin.x) * brickSizeInv.x) * poolSampleRegionSize.x;
        voxelAddress.y = poolOffset.y + ((p.y - brickBegin.y) * brickSizeInv.y) * poolSampleRegionSize.y;
        voxelAddress.z = poolOffset.z + ((p.z - brickBegin.z) * brickSizeInv.z) * poolSampleRegionSize.z;

        /* Sample the volume */
        float value = tex3D(volumeTexture, voxelAddress.x, voxelAddress.y, voxelAddress.z);

        /* Tranform from voxel intesity to colour */
        vec4f colour = tex1D(transferFunction, value);

        /* Apply opacity correction and accumulate colour */
        colour.w = 1.0f - powf(1.0f - colour.w, opacityCorrection);
        float opacity = 1.0f - a.w;
        colour.w *= opacity;
        a.x = colour.x * colour.w + a.x;
        a.y = colour.y * colour.w + a.y;
        a.z = colour.z * colour.w + a.z;
        a.w = colour.w + a.w;

        /* Step along the ray */
        p += step;
    }
    prd.out.pageTableAccesses += ptaccesses;

    accumulation = a;
}


/*! optix program for entering a volume region */
RT_PROGRAM void closest_hit()
{
    if(!dontSample)
    {
        accumulate(
            ray,
            prd.hit.entryDistance,
            prd.hit.exitDistance,
            prd.out.accumulation
        );
    }
}