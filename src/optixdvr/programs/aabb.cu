#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include "prd.h"

rtBuffer<float4> aabbMinBuffer;
rtBuffer<float4> aabbMaxBuffer;
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData, prd, rtPayload, );

RT_PROGRAM void get_aabb_bounds(int pid, float result[6])
{
    result[0] = aabbMinBuffer[pid].x;
    result[1] = aabbMinBuffer[pid].y;
    result[2] = aabbMinBuffer[pid].z;
    result[3] = aabbMaxBuffer[pid].x;
    result[4] = aabbMaxBuffer[pid].y;
    result[5] = aabbMaxBuffer[pid].z;
}

RT_PROGRAM void hit_aabb(int pid)
{
    float3 aabbMin = make_float3(aabbMinBuffer[pid].x, aabbMinBuffer[pid].y, aabbMinBuffer[pid].z);
    float3 aabbMax = make_float3(aabbMaxBuffer[pid].x, aabbMaxBuffer[pid].y, aabbMaxBuffer[pid].z);
    float3 vminv = (aabbMin - ray.origin) * prd.in.rayDirectionInverse;
    float3 vmaxv = (aabbMax - ray.origin) * prd.in.rayDirectionInverse;
    float3 tnear = fminf(vminv, vmaxv);
    float3 tfar = fmaxf(vminv, vmaxv);
    float tmin = fmaxf(tnear);
    float tmax = fminf(tfar);

    tmin = fmaxf(ray.tmin, tmin);
    if(tmax > tmin)
    {
        if(rtPotentialIntersection(tmin))
        {
            prd.hit.entryDistance = tmin;
            prd.hit.exitDistance = tmax;
            rtReportIntersection(0);
        }
    }
}