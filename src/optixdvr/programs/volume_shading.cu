#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "prd.h"

/*! the parameters that describe each individual sphere geometry */
// rtDeclareVariable(float3, boxMin, , );
// rtDeclareVariable(float3, boxMax, , );
rtDeclareVariable(float3, poolRegionOffset, , );
rtDeclareVariable(float3, poolRegionSize, , );

/*! the implicit state's ray we will intersect against */
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

/*! the attributes we use to communicate between intersection programs and hit program */
rtDeclareVariable(float3, hit_rec_normal, attribute hit_rec_normal, );
rtDeclareVariable(float3, hit_rec_p, attribute hit_rec_p, );

/*! the per ray data we operate on */
rtDeclareVariable(PerRayData, prd, rtPayload, );

rtDeclareVariable(float, ertThreshold, , );
rtTextureSampler<float, 3> volumeTexture;
rtTextureSampler<float4, 1> transferFunction;
rtDeclareVariable(float3, volumeDimensions, ,  );
rtDeclareVariable(float3, lightPositionVolumeSpace, , );
rtDeclareVariable(float3, gradientStep, , );


inline __device__ void accumulate(
    const optix::Ray &ray_in,
    float entryDistance,
    float exitDistance,
    vec4f &accumulation)
{
    // const vec3f rayDirection = vec3f(ray_in.direction);
    // const vec3f rayOrigin = vec3f(ray_in.origin);
    // entryDistance = fmaxf(prd.in.worldSpaceStepSize, entryDistance);
    // exitDistance = fmaxf(prd.in.worldSpaceStepSize, exitDistance);
    // const float depth = exitDistance - entryDistance;

    // vec3f worldSpaceEntry;
    // worldSpaceEntry.x = rayDirection.x * entryDistance + rayOrigin.x;
    // worldSpaceEntry.y = rayDirection.y * entryDistance + rayOrigin.y;
    // worldSpaceEntry.z = rayDirection.z * entryDistance + rayOrigin.z;

    // const int steps = depth / prd.in.worldSpaceStepSize;
    // const vec3f brickEntryPoint = (worldSpaceEntry - prd.hit.boxMin) / (prd.hit.boxMax - prd.hit.boxMin);
    // const vec3f step = prd.in.poolSpaceStep;

    // /* Ray-stepping loop */
    // vec3f p;
    // p.x = brickEntryPoint.x * poolRegionSize.x + poolRegionOffset.x;
    // p.y = brickEntryPoint.y * poolRegionSize.y + poolRegionOffset.y;
    // p.z = brickEntryPoint.z * poolRegionSize.z + poolRegionOffset.z;

    // vec4f a = accumulation;
    // const float opacityCorrection = prd.in.opacityCorrection;
    // for(int i = 0; i < steps && a.w < ertThreshold; ++i)
    // {
    //     float value = tex3D(volumeTexture, p.x, p.y, p.z);
    //     vec4f colour = tex1D(transferFunction, value);

    //     if(colour.w > 0.0f)
    //     {
    //         vec3f normal;
    //         normal.x = tex3D(volumeTexture, p.x + gradientStep.x, p.y, p.z)
    //                 - tex3D(volumeTexture, p.x - gradientStep.x, p.y, p.z);
    //         normal.y = tex3D(volumeTexture, p.x, p.y + gradientStep.y, p.z)
    //                 - tex3D(volumeTexture, p.x, p.y - gradientStep.y, p.z);
    //         normal.z = tex3D(volumeTexture, p.x, p.y, p.z + gradientStep.z)
    //                 - tex3D(volumeTexture, p.x, p.y, p.z - gradientStep.z);
    //         normal = normalize(normal);

    //         vec3f lightPosition(2, 0, 0);
    //         vec3f toLight = normalize(lightPosition - p);
    //         float diff = fmaxf(fminf(dot(toLight, normal), 1.0f), 0.0f);

    //         colour.x *= diff;
    //         colour.y *= diff;
    //         colour.z *= diff;
    //         float opacity = 1.0f - a.w;
    //         a += colour * opacity;
    //     }


    //     p += step;
    // }

    // accumulation = a;
}


/*! optix program for entering a volume region */
RT_PROGRAM void closest_hit()
{
    accumulate(
        ray,
        prd.hit.entryDistance,
        prd.hit.exitDistance,
        prd.out.accumulation
    );
}